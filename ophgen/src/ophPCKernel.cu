#include "hip/hip_runtime.h"
#ifndef OphPCKernel_cu__
#define OphPCKernel_cu__

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/device_functions.h>

#include "typedef.h"
#include "ophPointCloud_GPU.h"


__global__ void cudaKernel_diffractEncodedRS(Real* pc_data, Real* amp_data, const GpuConstERS* config, const int n_points_stream, Real* dst) {
	ulonglong tid = blockIdx.x * blockDim.x + threadIdx.x;
	ulonglong tid_offset = blockDim.x * gridDim.x;
	ulonglong n_pixels = config->pn_X * config->pn_Y;

	for (tid; tid < n_pixels; tid += tid_offset) {
		int xxtr = tid % config->pn_X;
		int yytr = tid / config->pn_X;
		ulonglong idx = xxtr + yytr * config->pn_X;

		Real xxx = ((Real)xxtr + 0.5) * config->pp_X - config->half_ss_X;
		Real yyy = config->half_ss_Y - ((Real)yytr + 0.5) * config->pp_Y;
		Real interWav = xxx * config->sin_thetaX + yyy * config->sin_thetaY;

		for (int j = 0; j < n_points_stream; ++j) { //Create Fringe Pattern
			Real pcx = pc_data[3 * j + _X] * config->scale_X;
			Real pcy = pc_data[3 * j + _Y] * config->scale_Y;
			Real pcz = pc_data[3 * j + _Z] * config->scale_Z + config->offset_depth;

			Real r = sqrt((xxx - pcx) * (xxx - pcx) + (yyy - pcy) * (yyy - pcy) + (pcz * pcz));
			Real p = config->k * (r - interWav);
			Real res = amp_data[config->n_colors * j] * cos(p);

			*(dst + idx) += res;
		}
	}
	__syncthreads();
}


__global__ void cudaKernel_diffractNotEncodedRS(Real* pc_data, Real* amp_data, const GpuConstNERS* config, const int n_points_stream, Real* dst_real, Real* dst_imag) {
	ulonglong tid = blockIdx.x * blockDim.x + threadIdx.x;
	ulonglong tid_offset = blockDim.x * gridDim.x;
	ulonglong n_pixels = config->pn_X * config->pn_Y;

	for (tid; tid < n_pixels; tid += tid_offset) {
		int xxtr = tid % config->pn_X;
		int yytr = tid / config->pn_X;
		ulonglong idx = xxtr + yytr * config->pn_X;

		Real xxx = -config->half_ss_X + (xxtr - 1) * config->pp_X;
		Real yyy = -config->half_ss_Y + (config->pn_Y - yytr) * config->pp_Y;

		for (int j = 0; j < n_points_stream; ++j) { //Create Fringe Pattern
			Real pcx = pc_data[3 * j + _X] * config->scale_X;
			Real pcy = pc_data[3 * j + _Y] * config->scale_Y;
			Real pcz = pc_data[3 * j + _Z] * config->scale_Z + config->offset_depth;
			Real amplitude = amp_data[config->n_colors * j];

			//boundary test
			Real abs_det_txy_pcz = abs(config->det_tx * pcz);
			Real _xbound[2] = {
				pcx + abs_det_txy_pcz,
				pcx - abs_det_txy_pcz
			};

			abs_det_txy_pcz = abs(config->det_ty * pcz);
			Real _ybound[2] = {
				pcy + abs_det_txy_pcz,
				pcy - abs_det_txy_pcz
			};

			Real Xbound[2] = {
				floor((_xbound[0] + config->half_ss_X) / config->pp_X) + 1,
				floor((_xbound[1] + config->half_ss_X) / config->pp_X) + 1
			};

			Real Ybound[2] = {
				config->pn_Y - floor((_ybound[1] + config->half_ss_Y) / config->pp_Y),
				config->pn_Y - floor((_ybound[0] + config->half_ss_Y) / config->pp_Y)
			};

			if (Xbound[0] > config->pn_X)	Xbound[0] = config->pn_X;
			if (Xbound[1] < 0)				Xbound[1] = 0;
			if (Ybound[0] > config->pn_Y)	Ybound[0] = config->pn_Y;
			if (Ybound[1] < 0)				Ybound[1] = 0;
			//

			if (((xxtr >= Xbound[1]) && (xxtr < Xbound[0])) && ((yytr >= Ybound[1]) && (yytr < Ybound[0]))) {
				Real xxx_pcx_sq = (xxx - pcx) * (xxx - pcx);
				Real yyy_pcy_sq = (yyy - pcy) * (yyy - pcy);
				Real pcz_sq = pcz * pcz;

				//range test
				Real abs_det_txy_sqrt = abs(config->det_tx * sqrt(yyy_pcy_sq + pcz_sq));
				Real range_x[2] = {
					pcx + abs_det_txy_sqrt,
					pcx - abs_det_txy_sqrt
				};

				abs_det_txy_sqrt = abs(config->det_ty * sqrt(xxx_pcx_sq + pcz_sq));
				Real range_y[2] = {
					pcy + abs_det_txy_sqrt,
					pcy - abs_det_txy_sqrt
				};
				//

				if (((xxx < range_x[0]) && (xxx > range_x[1])) && ((yyy < range_y[0]) && (yyy > range_y[1]))) {
					Real r = sqrt(xxx_pcx_sq + yyy_pcy_sq + pcz_sq);
					Real p = config->k * r;
					Real a = (amplitude * pcz) / (config->lambda * r * r);;
					Real res_real = sin(p) * a;
					Real res_imag = -cos(p) * a;

					*(dst_real + idx) += res_real;
					*(dst_imag + idx) += res_imag;
				}
			}
		}
	}
	__syncthreads();
}


__global__ void cudaKernel_diffractNotEncodedFrsn(Real* pc_data, Real* amp_data, const GpuConstNEFR* config, const int n_points_stream, Real* dst_real, Real* dst_imag) {
	ulonglong tid = blockIdx.x * blockDim.x + threadIdx.x;
	ulonglong tid_offset = blockDim.x * gridDim.x;
	ulonglong n_pixels = config->pn_X * config->pn_Y;

	for (tid; tid < n_pixels; tid += tid_offset) {
		int xxtr = tid % config->pn_X;
		int yytr = tid / config->pn_X;
		ulonglong idx = xxtr + yytr * config->pn_X;

		Real xxx = -config->half_ss_X + (xxtr - 1) * config->pp_X;
		Real yyy = -config->half_ss_Y + (config->pn_Y - yytr) * config->pp_Y;

		for (int j = 0; j < n_points_stream; ++j) { //Create Fringe Pattern
			Real pcx = pc_data[3 * j + _X] * config->scale_X;
			Real pcy = pc_data[3 * j + _Y] * config->scale_Y;
			Real pcz = pc_data[3 * j + _Z] * config->scale_Z + config->offset_depth;
			Real amplitude = amp_data[config->n_colors * j];

			//boundary test
			Real abs_txy_pcz = abs(config->tx * pcz);
			Real _xbound[2] = {
				pcx + abs_txy_pcz,
				pcx - abs_txy_pcz
			};

			abs_txy_pcz = abs(config->ty * pcz);
			Real _ybound[2] = {
				pcy + abs_txy_pcz,
				pcy - abs_txy_pcz
			};

			Real Xbound[2] = {
				floor((_xbound[0] + config->half_ss_X) / config->pp_X) + 1,
				floor((_xbound[1] + config->half_ss_X) / config->pp_X) + 1
			};

			Real Ybound[2] = {
				config->pn_Y - floor((_ybound[1] + config->half_ss_Y) / config->pp_Y),
				config->pn_Y - floor((_ybound[0] + config->half_ss_Y) / config->pp_Y)
			};

			if (Xbound[0] > config->pn_X)	Xbound[0] = config->pn_X;
			if (Xbound[1] < 0)				Xbound[1] = 0;
			if (Ybound[0] > config->pn_Y)	Ybound[0] = config->pn_Y;
			if (Ybound[1] < 0)				Ybound[1] = 0;
			//

			if (((xxtr >= Xbound[1]) && (xxtr < Xbound[0])) && ((yytr >= Ybound[1]) && (yytr < Ybound[0]))) {
				Real p = config->k * ((xxx - pcx) * (xxx - pcx) + (yyy - pcy) * (yyy - pcy) + (2 * pcz * pcz)) / (2 * pcz);
				Real a = amplitude / (config->lambda * pcz);
				Real res_real = sin(p) * a;
				Real res_imag = -cos(p) * a;

				*(dst_real + idx) += res_real;
				*(dst_imag + idx) += res_imag;
			}
		}
	}
	__syncthreads();
}


extern "C"
{
	void cudaGenCghPointCloud_EncodedRS(
		const int &nBlocks, const int &nThreads, const int &n_pts_per_stream,
		Real* cuda_pc_data, Real* cuda_amp_data,
		Real* cuda_dst,
		const GpuConstERS* cuda_config)
	{
		cudaKernel_diffractEncodedRS << < nBlocks, nThreads >> > (cuda_pc_data, cuda_amp_data, cuda_config, n_pts_per_stream, cuda_dst);
	}

	void cudaGenCghPointCloud_NotEncodedRS(
		const int &nBlocks, const int &nThreads, const int &n_pts_per_stream,
		Real* cuda_pc_data, Real* cuda_amp_data,
		Real* cuda_dst_real, Real* cuda_dst_imag,
		const GpuConstNERS* cuda_config)
	{
		cudaKernel_diffractNotEncodedRS << < nBlocks, nThreads >> > (cuda_pc_data, cuda_amp_data, cuda_config, n_pts_per_stream, cuda_dst_real, cuda_dst_imag);
	}

	void cudaGenCghPointCloud_NotEncodedFrsn(
		const int &nBlocks, const int &nThreads, const int &n_pts_per_stream,
		Real* cuda_pc_data, Real* cuda_amp_data,
		Real* cuda_dst_real, Real* cuda_dst_imag,
		const GpuConstNEFR* cuda_config)
	{
		cudaKernel_diffractNotEncodedFrsn << < nBlocks, nThreads >> > (cuda_pc_data, cuda_amp_data, cuda_config, n_pts_per_stream, cuda_dst_real, cuda_dst_imag);
	}
}

#endif // !OphPCKernel_cu__